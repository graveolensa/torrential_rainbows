#include "hip/hip_runtime.h"
#include "kernel.h"
#define TX 32
#define TY 32

#define DIM 2100

struct hipComplex {
    float   r;
    float   i;
    __device__ hipComplex( float a, float b ) : r(a), i(b)  {}
    __device__ float magnitude2( void ) {
        return r * r + i * i;
    }
    __device__ hipComplex operator*(const hipComplex& a) {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    __device__ hipComplex operator-(const hipComplex& a) {
        return hipComplex(r-a.r, i-a.i);
    }
    __device__ hipComplex operator+(const hipComplex& a) {
        return hipComplex(r+a.r, i+a.i);
    }
    __device__ hipComplex operator/(const hipComplex& a) {
        return hipComplex((r*a.r + i*a.i)/(a.r*a.r + a.i*a.i), (i*a.r - r*a.i)/(a.r*a.r + a.i*a.i));
    }
};


__device__ hipComplex conj(hipComplex m)
{
    hipComplex out(m.r,-m.i);
    return out;
}


__device__ hipComplex nor(hipComplex m)
{
	hipComplex out(m.r*m.r+m.i*m.i,0.0);
	return out;
}

__device__ float  norg(hipComplex m)
{
    return sqrtf(m.r*m.r+m.i*m.i);
}




__device__ hipComplex qpoch(hipComplex a, hipComplex q) {
    hipComplex out(1.0,0.0);
    hipComplex unity(1.0,0.0);
    int i = 0;
    hipComplex Q = q;
    if(q.magnitude2()>1.0)
    {
        return hipComplex(0.0,0.0);
    }
    // We want to formally match the definition of a q-pochhammer symbol.
    for(i=1;i<80;i++)
    {
        out = out * (unity - a*Q);
        Q = q * Q;
    }
    return out;  
}

__device__ hipComplex qp(hipComplex a, hipComplex q, int n) {
    hipComplex out(1.0,0.0);
    hipComplex unity(1.0,0.0);
    int i = 0;
    hipComplex Q = q;
    if(q.magnitude2()>1.0)
    {
        return hipComplex(0.0,0.0);
    }
    // We want to formally match the definition of a q-pochhammer symbol.
    for(i=1;i<n;i++)
    {
        out = out * (unity - a*Q);
        Q = q * Q;
    }
    return out;
}

__device__ hipComplex ramphi(hipComplex q) {
    hipComplex out(1.0,0.0);
    hipComplex mone(-1.0,0.0);
    hipComplex mq = mone*q;
    return qpoch(mq,mq)/qpoch(q,mq);
}

__device__ hipComplex rampsi(hipComplex q) {
    hipComplex out(1.0,0.0);
    hipComplex mone(-1.0,0.0);
    hipComplex mq = mone*q;
    return qpoch(mq,q)*qpoch(q*q,q*q);
}

__device__ hipComplex ramchi(hipComplex q) {
    hipComplex out(1.0,0.0);
    hipComplex mone(-1.0,0.0);
    hipComplex mq = mone*q;
    return qpoch(mq,q*q);
}

__device__ hipComplex ramf(hipComplex a, hipComplex b) {
    hipComplex out(1.0,0.0);
    hipComplex mone(-1.0,0.0);
    hipComplex ma = mone*a;
    hipComplex mb = mone*b;
    return qpoch(ma,a*b)*qpoch(mb,a*b)*qpoch(a*b,a*b);
}






// complex exponential
__device__ hipComplex expc(hipComplex m)
{
  hipComplex out(expf(m.r) * cosf(m.i),expf(m.r) * sinf(m.i));
  return out;
}



__device__ hipComplex powc(hipComplex ag, hipComplex bg)
{  
  hipComplex out(0.0,0.0);
  hipComplex mesp(0.0,0.0);
  hipComplex frim(0.0,0.0);
  double radiu, thet;
  /* get the proper polar form of the complex number */
  radiu =  sqrtf(ag.r*ag.r + ag.i*ag.i);
  thet = atan2f(ag.i,ag.r);
  /* mesp gives R^(c+di) */
  mesp.r = powf(radiu,bg.r)*cosf(bg.i*logf(radiu));
  mesp.i = powf(radiu,bg.r)*sinf(bg.i*logf(radiu));
  /* frim gives e^(i theta (c+di)) */
  /* now since we already have the machinery
     for performing complex exponentiation (just exp), we
     can just call that here */
  frim.r = -1.0 * bg.i * thet;
  frim.i = bg.r * thet;
  frim = expc(frim);  
  out = mesp*frim;
  return out;
}


// cosine (nothing algorithmically clean)
__device__ hipComplex cosc(hipComplex m)
{
    hipComplex ai(0.0,1.0);
    hipComplex ot(0.5,0.0);
    hipComplex mone(-1.0,0.0);
    hipComplex out = ot*(expc(m*ai) + expc(mone*m*ai));
    return out;
}

__device__ hipComplex sins(hipComplex m)
{
    hipComplex ai(0.0,1.0);
    hipComplex ot(0.0,0.5);
    hipComplex mone(-1.0,0.0);
    hipComplex out = ot*(expc(m*ai) - expc(mone*m*ai));
    return out;
}

__device__ hipComplex tans(hipComplex m)
{
    return sins(m)/cosc(m);
}

__device__ hipComplex moeb(hipComplex t, hipComplex a, hipComplex z)
{
    hipComplex out(0.0,0.0);
    hipComplex ai(0.0,1.0);
    hipComplex unity(1.0,0.0);
    out = expc(ai*t) * (z-a)/(unity-conj(a)*z);
    return out;
}

__device__ hipComplex mob(hipComplex a, hipComplex z)
{
    hipComplex out(0.0,0.0);
    hipComplex ai(0.0,1.0);
    hipComplex unity(1.0,0.0);
    out =  (z-a)/(unity-conj(a)*z);
    return out;
}


__device__ hipComplex bnewt(hipComplex z) {
    hipComplex three(3.0,0.0);
    hipComplex unity(1.0,0.0);
    hipComplex out(0.0,0.0);
    hipComplex Z =z;
    hipComplex L(0.0,0.0);
    
    hipComplex R(0.62348980185873359,0.7818314824680298);
    hipComplex v(0.62348980185873359,0.7818314824680298);
    int i;
    for(i=0;i<100;i++)
    {
        L = sins(expc(Z)-cosc(Z))-Z;
        out = out + v*L;
        v = R * v;
        Z = Z - L/((expc(Z)+sins(Z))*cosc(expc(Z)-cosc(Z))-unity);
    }
    return out;
}

__device__ hipComplex they3(hipComplex z, hipComplex q)
{
    int u;
    hipComplex out(0.0,0.0);
    hipComplex enn(-20.0,0.0);
    hipComplex onn(1.0,0.0);
    hipComplex dui(0.0,1.0);
    for(u=-20;u<20;u++)
    {
        out = out + powc(q,enn*enn)*expc(dui*enn*z);
        enn = enn + onn;
    }
    return out;
}


__device__ hipComplex  wahi(hipComplex z)
{
    int u;
    hipComplex un(1.0,0.0);
    hipComplex ne(1.0,0.0);
 hipComplex out(0.0,0.0);
 for(u=1;u<40;u++)
 {
 	out = out + powc(z/ne,ne);
 	ne = ne + un;
 }
 out = out + un;
 return out;
}

__device__ hipComplex  dwahi(hipComplex z)
{
    int u;
    hipComplex un(1.0,0.0);
    hipComplex ne(1.0,0.0);
 hipComplex out(0.0,0.0);
 for(u=1;u<40;u++)
 {
 	out = out + powc(z/ne,ne-un);
 	ne = ne + un;
 }
 return out;
}


__device__ hipComplex they3p(hipComplex z, hipComplex q)
{
    int u;
    hipComplex out(0.0,0.0);
    hipComplex enn(-20.0,0.0);
    hipComplex onn(1.0,0.0);
    hipComplex dui(0.0,1.0);
    for(u=-20;u<20;u++)
    {
        out = out + (enn*enn)*powc(q,enn*enn-onn)*expc(dui*enn*z);
        enn = enn + onn;
    }
    return out;
}

__device__ hipComplex h3ey3p(hipComplex z, hipComplex q)
{
    int u;
    hipComplex out(0.0,0.0);
    hipComplex aut(0.0,0.0);
    hipComplex enn(-20.0,0.0);
    hipComplex onn(1.0,0.0);
    hipComplex dui(0.0,1.0);
    hipComplex vel(0.0,0.0);
    hipComplex rav(0.0,0.0);
    for(u=-40;u<40;u++)
    {
        vel = expc(dui*enn*z);
        rav = powc(q,enn*enn);
        aut = aut + (enn*enn)*rav/q*vel;
        out = out + rav*vel;
        enn = enn + onn;
    }
    return out/aut;
}


__device__ hipComplex thess(hipComplex z, hipComplex q)
{
	int v;
	hipComplex unity(1.0,0.0);
	hipComplex out(1.0,0.0);
	hipComplex tw(2.0,0.0);
	hipComplex qoo(1.0,0.0);
	 for(v=0;v<10;v++)
    {
        qoo = qoo * q * q;
        out = out * (unity - qoo) * (unity + tw * qoo/q * cosc(tw*z) + qoo*qoo/(q*q)); 
        
    }
    return out;
}






__device__ hipComplex the1(hipComplex z, hipComplex q)
{
    int v;
    hipComplex unity(1.0,0.0);
    hipComplex out(1.0,0.0);
    hipComplex tw(2.0,0.0);
    hipComplex qoo(1.0,0.0);
     hipComplex rt(0.25,0.0);
     for(v=0;v<10;v++)
    {
        qoo = qoo * q * q;
        out = out * (unity - qoo) * (unity + tw * qoo/q * cosc(tw*z) + qoo*qoo/(q*q)); 
        
    }
    return tw*out*powc(q,rt)*sins(z);
}

__device__ hipComplex the2(hipComplex z, hipComplex q)
{
    int v;
    hipComplex unity(1.0,0.0);
    hipComplex out(1.0,0.0);
    hipComplex tw(2.0,0.0);
    hipComplex qoo(1.0,0.0);
     hipComplex rt(0.25,0.0);
     for(v=0;v<10;v++)
    {
        qoo = qoo * q * q;
        out = out * (unity - qoo) * (unity - tw * qoo/q * cosc(tw*z) + qoo*qoo/(q*q)); 
        
    }
    return tw*out*powc(q,rt)*cosc(z);
}

__device__ hipComplex the3(hipComplex z, hipComplex q)
{
    int v;
    hipComplex unity(1.0,0.0);
    hipComplex out(1.0,0.0);
    hipComplex tw(2.0,0.0);
    hipComplex qoo(1.0,0.0);
     for(v=0;v<10;v++)
    {
        qoo = qoo * q * q;
        out = out * (unity - qoo) * (unity + tw * qoo/q * cosc(tw*z) + qoo*qoo/(q*q)); 
        
    }
    return out;
}


__device__ hipComplex the4(hipComplex z, hipComplex q)
{
    int v;
    hipComplex unity(1.0,0.0);
    hipComplex out(1.0,0.0);
    hipComplex tw(2.0,0.0);
    hipComplex qoo(1.0,0.0);
     for(v=0;v<10;v++)
    {
        qoo = qoo * q * q;
        out = out * (unity - qoo) * (unity - tw * qoo/q * cosc(tw*z) + qoo*qoo/(q*q)); 
        
    }
    return out;
}

/* routine to generate q-integers */
__device__ hipComplex qin(hipComplex a, hipComplex q)
{
    hipComplex unity(1.0,0.0);
    hipComplex out(1.0,0.0);
    out = (unity - powc(q, a))/(unity-q);
    return out;
}

/* generating function for n^2 */
__device__ hipComplex geffa(hipComplex z, hipComplex q)
{
    hipComplex out(0.0,0.0);
    hipComplex unity(1.0,0.0);
    hipComplex wu(0.0,0.0);
    hipComplex Z=unity;
    int v;
    for(v=0;v<20;v++)
    {
        out = out +  qin(wu*wu,q)* Z;
        wu = wu + unity; 
        Z = z * Z;
    }
return out;
}








__device__ hipComplex thratd(hipComplex z, hipComplex q)
{
	int n;
	hipComplex fau(4.0,0.0);
	hipComplex too(2.0,0.0);
	hipComplex unity(1.0,0.0);
	hipComplex ennn(1.0,0.0);
	hipComplex ni(-1.0,0.0);
	hipComplex noo(-1.0,0.0);
	hipComplex out(0.0,0.0);
	hipComplex loo = q;
	hipComplex qoo =q*q;
	for(n=0;n<80;n++)
	{
		out = out + noo*(loo/(unity-qoo))*sins(too*ennn*z);
		qoo = qoo * q*q;
		loo = loo * q;
		ennn = ennn +unity;
		noo = ni * noo;
	}
	return out*fau;
}

__device__ hipComplex thess4(hipComplex z, hipComplex q)
{
	int v;
	hipComplex unity(1.0,0.0);
	hipComplex out(1.0,0.0);
	hipComplex tw(2.0,0.0);
	hipComplex qoo(1.0,0.0);
	 for(v=0;v<20;v++)
    {
        qoo = qoo * q * q;
        out = out * (unity - qoo) * (unity - tw * qoo/q * cosc(tw*z) + qoo*qoo/(q*q)); 
        
    }
    return out;
}




__device__ hipComplex thesk(hipComplex z, hipComplex q, hipComplex r)
{
    int v;
    hipComplex unity(1.0,0.0);
    hipComplex out(1.0,0.0);
    hipComplex tw(2.0,0.0);
    hipComplex qoo(1.0,0.0);
    hipComplex roo(1.0,0.0);
     for(v=0;v<20;v++)
    {
        qoo = qoo * q * q;
        roo = roo *  r * r ;
        out = out * (unity - qoo) * (unity + tw * qoo/q * cosc(tw*z) + roo*roo/(r*r)); 
        
    }
    return out;
}



__device__ hipComplex thass(hipComplex z, hipComplex q)
{
    int v;
    hipComplex unity(1.0,0.0);
    hipComplex out(1.0,0.0);
    hipComplex tw(2.0,0.0);
    hipComplex qoo(1.0,0.0);
     for(v=0;v<20;v++)
    {
        qoo = qoo * q * q;
        out = out * (unity - qoo) * (unity + tw * qoo/q * sins(tw*z) + qoo*qoo/(q*q)); 
        
    }
    return out;
}

__device__ hipComplex rogers( hipComplex q)
{
    hipComplex onf(0.2,0.0);
    hipComplex Q5 = q*q*q*q*q;
    hipComplex out = powc(q,onf)* qpoch(q,Q5) * qpoch(q*q*q*q,Q5)/ (qpoch(q*q,Q5)*qpoch(q*q*q,Q5));
    return out;
}

__device__ hipComplex flat(hipComplex m)
{
    float ua = sqrtf(m.r*m.r + m.i*m.i);
    hipComplex out(m.r/ua,m.i/ua);
    return out;
}

__device__ hipComplex eff(hipComplex z, hipComplex lambda)
{
    return z*z*z*z+ lambda/(z*z*z*z);
}

__device__  hipComplex thete(float R, hipComplex tau, hipComplex z)
{
    /* note that as I'm not immediately doing this on the unit circle, as the real
    action is considered to happen on the z-plane, we don't yet need to fret about
    whether I'm looking at things in terms of tau or in terms of q, next revision */
   /* set accumulant to zero */
    hipComplex A(0.0,0.0);
    /* miscellaneous setup */
    hipComplex pai(3.14159265353898,0.0);
    hipComplex ai(0.0,1.0);
    hipComplex oo(1.0,0.0);
    hipComplex oot(2.0,0.0);
    hipComplex nini(9.0,0.0);
    hipComplex eigh(-18.0,0.0);
    /* hipComplex arr(cos(2*3.1415926535897f*R/2048.0),0.0) */
    hipComplex frann(1.0,0.0);
    frann = pai * ai * tau ;
    hipComplex shenn(1.0,0.0);
    shenn = oot * ai * z;
    hipComplex plenn(1.0,0.0);
    hipComplex enn(1.0,0.0);
    hipComplex ann(1.0,0.0);
    hipComplex bnn(1.0,0.0);
    hipComplex scrunn(1.0,0.0);
    float ca, cb,cc;
    int a,  b;
    for(a=-10;a<10;a++)
    {
        ann.r = a;
        for(b=-10;b<10;b++)
        {
                bnn.r = b;
                if(((a+b)%2)==0)
                {
                        scrunn.r = a*a + b*b;
                        A = A + expc(frann* scrunn) * expc(shenn* (ann+bnn));
                }
                else
                {
                        
                        ca = 5.0 + a*a + b*b;
                        cb =  2*(a * cos(R)- b * sin(R));
                        cc  =  4*(b * cos(R)+a*sin(R));
                        scrunn.r = ca + cb + cc;
                        A = A + expc(frann*scrunn)*expc(shenn*(ann+bnn));
                }
        }
    }
    return A;
}
 
__device__  hipComplex thetta(hipComplex tau, hipComplex z)
{
    /* note that as I'm not immediately doing this on the unit circle, as the real
    action is considered to happen on the z-plane, we don't yet need to fret about
    whether I'm looking at things in terms of tau or in terms of q, next revision */
   /* set accumulant to zero */
    hipComplex A(0.0,0.0);
    /* miscellaneous setup */
    hipComplex pai(3.14159265353898,0.0);
    hipComplex ai(0.0,1.0);
    hipComplex oo(1.0,0.0);
    hipComplex oot(2.0,0.0);
    hipComplex nini(9.0,0.0);
    hipComplex eigh(-18.0,0.0);
    /* hipComplex arr(cos(2*3.1415926535897f*R/2048.0),0.0) */
    hipComplex frann(1.0,0.0);
    frann = pai * ai * tau ;
    hipComplex shenn(1.0,0.0);
    shenn = oot * ai * z;
    hipComplex plenn(1.0,0.0);
    hipComplex enn(1.0,0.0);
    int n;
    for(n=-10;n<10;n++)
    {
        enn.r = n;
        plenn = enn * enn;
        /* this get the hipComplex out of the event loop */
        A = A + expc(frann* plenn) * expc(shenn* enn);
}
return A;
}

__device__ hipComplex mitlef(hipComplex z,hipComplex c)
{
    hipComplex out(0.0,0.0);
    hipComplex Z(1.0,0.0);
    hipComplex frove(0.0,0.0);
    int v;
    for(v=0;v<20;v++)
    {
        frove.r = tgammaf(c.r*v+c.i);
        out = out + Z/frove;
        Z = Z * z;
    }
    return out;
}

__device__ hipComplex helva(hipComplex z)
{
    hipComplex out(j0f(z.r),j1f(z.i));
    return  out;
}

__device__ hipComplex hylva(hipComplex z)
{
    hipComplex out(j1f(1/j0f(z.r)),j1f(1/j0f(z.i)));
    return  out;
}


__device__ hipComplex hilva(hipComplex z)
{
    hipComplex out(j1f(z.r),j0f(z.i));
    return  out;
}


__device__ hipComplex halva(hipComplex z)
{
    hipComplex out(j0f(z.r),j0f(z.i));
    return  out;
}

__device__ hipComplex hinva(hipComplex z)
{
    hipComplex out(j1f(z.r),j1f(z.i));
    return  out;
}

__device__ hipComplex henga(hipComplex z)
{
    hipComplex out(acoshf(z.r),asinhf(z.i));
    return  out;
}

__device__ hipComplex holva(hipComplex z)
{
    hipComplex out(y0f(z.r),y1f(z.i));
    return  out;
}


__device__ hipComplex aliva(hipComplex z)
{
    hipComplex out(j1f(z.r),cyl_bessel_i1f(z.i));
    return  out;
}

__device__ hipComplex ariva(hipComplex z)
{
    hipComplex out(sinf(z.i),cbrtf(z.r));
    return  out;
}


__device__ hipComplex arago(hipComplex z, hipComplex q)
{
    int v;
    hipComplex unity(1.0,0.0);
    hipComplex out(1.0,0.0);
    hipComplex tw(2.0,0.0);
    hipComplex qoo(1.0,0.0);
     for(v=0;v<10;v++)
    {
        qoo = qoo * q * q;
        out = out * (unity - qoo) * (unity + tw * qoo/q * hinva(tw*z) + qoo*qoo/(q*q)); 
        
    }
    return out;
}


__device__ hipComplex irigo(hipComplex z, hipComplex q)
{
    int v;
    hipComplex unity(1.0,0.0);
    hipComplex out(1.0,0.0);
    hipComplex tw(2.0,0.0);
    hipComplex qoo(1.0,0.0);
     for(v=0;v<10;v++)
    {
        qoo = qoo * q * q;
        out = out * (unity - qoo) * (unity + tw * qoo/q * holva(tw*z) + qoo*qoo/(q*q)); 
        
    }
    return out;
}

__device__ hipComplex urigo(hipComplex z, hipComplex q)
{
    int v;
    hipComplex unity(1.0,0.0);
    hipComplex out(1.0,0.0);
    hipComplex tw(2.0,0.0);
    hipComplex qoo(1.0,0.0);
     for(v=0;v<10;v++)
    {
        qoo = qoo * q * q;
        out = out * (unity - qoo) * (unity + tw * qoo/q * powc(hilva(q*z),helva(q*z)) + qoo*qoo/(q*q)); 
        
    }
    return out;
}


__device__ hipComplex arreg(hipComplex q, hipComplex r, hipComplex z)
{
	/* arreg implements the representation of theta3'(z)/theta(z) I don't know if these are
		derivatives with respect to z or q, we'll see */
	hipComplex out(0.0,0.0);
	hipComplex qoo(1.0,0.0);
	hipComplex roo(1.0,0.0);
	hipComplex morra(-1.0,0.0);
	hipComplex tla(1.0,0.0);
	hipComplex vnn(0.0,0.0);
	hipComplex fou(4.0,0.0);
	hipComplex tw(2.0,0.0);
	hipComplex run(1.0,0.0);
	int v;
	for(v=0;v<20;v++)
	{
		qoo = qoo * q;
		roo = roo * r * r;
		tla = tla * morra;
		vnn = vnn + run;
		out = out + morra*qoo*sins(tw*z*run)/(run-roo);
	}
return fou*out;
}

__device__ hipComplex urreg(hipComplex q, hipComplex r, hipComplex z)
{
	/* arreg implements the representation of theta3'(z)/theta(z) I don't know if these are
		derivatives with respect to z or q, we'll see */
	hipComplex out(0.0,0.0);
	hipComplex qoo(1.0,0.0);
	hipComplex roo(1.0,0.0);
	hipComplex morra(-1.0,0.0);
	hipComplex tla(1.0,0.0);
	hipComplex vnn(0.0,0.0);
	hipComplex fou(4.0,0.0);
	hipComplex tw(2.0,0.0);
	hipComplex run(1.0,0.0);
	int v;
	for(v=0;v<10;v++)
	{
		qoo = qoo * q;
		roo = roo * r * r;
		tla = tla * morra;
		vnn = vnn + run;
		out = out + morra*qoo*the3(tw*z*run,r)/(run-roo);
	}
return fou*out;
}


// * small q-exponential
__device__ hipComplex qexp(hipComplex z, hipComplex q)
{
	hipComplex mone(-1.0,0.0);
	hipComplex une(1.0,0.0);
	return une/qpoch(z,q);
}

//* large q exponential is just qpoch(-z,q)
__device__ hipComplex qExp(hipComplex z, hipComplex q)
{
	hipComplex mone(-1.0,0.0);
	hipComplex une(1.0,0.0);
	return qpoch(mone*z,q);
}




__device__ hipComplex sinq(hipComplex z, hipComplex q)
{
	hipComplex aie(0.0,1.0);
	hipComplex out(0.0,0.0);
	hipComplex doo(2.0,0.0);
	out = (qexp(z*aie,q) -qexp(z*aie,q))/doo;
	return out;
}


__device__ hipComplex cosq(hipComplex z, hipComplex q)
{
	hipComplex aie(0.0,1.0);
	hipComplex out(0.0,0.0);
	hipComplex doo(2.0,0.0);
	out = (qexp(z*aie,q) +qexp(z*aie,q))/doo;
	return out;
}

__device__ hipComplex Sinq(hipComplex z, hipComplex q)
{
	hipComplex aie(0.0,1.0);
	hipComplex out(0.0,0.0);
	hipComplex doo(2.0,0.0);
	out = (qExp(z*aie,q) -qExp(z*aie,q))/doo;
	return out;
}


__device__ hipComplex Cosq(hipComplex z, hipComplex q)
{
	hipComplex aie(0.0,1.0);
	hipComplex out(0.0,0.0);
	hipComplex doo(2.0,0.0);
	out = (qExp(z*aie,q) +qExp(z*aie,q))/doo;
	return out;
}

__device__ hipComplex asins(hipComplex z)
{
	float alp = 0.5 * (sqrtf((z.r+1)*(z.r+1) + z.i*z.i) + sqrtf((z.r-1)*(z.r-1) + z.i*z.i));
	float bet = 0.5 * (sqrtf((z.r+1)*(z.r+1) + z.i*z.i) - sqrtf((z.r-1)*(z.r-1) + z.i*z.i));
	float fla = z.i/abs(z.i);
	// *signum, but without a comparison, probably a saner way to do this? //
	hipComplex out(0.0,0.0);
	out.r = asinf(bet);
	out.i = fla * logf(alp + sqrtf(alp*alp-1));
	return out;
}


__device__ int gcd(int a, int b) 
{
    int remainder = a % b;
    if (remainder == 0) {
        return b;
    }
    return gcd(b, remainder);
}




/* Real Analytic Eisenstein Series */

__device__ hipComplex reis(hipComplex s, hipComplex z)
{
	// see en.wikipedia.org/wiki/Real_analytic_Eisenstein_series
	hipComplex out(0.0,0.0);
	hipComplex hav(0.5,0.0);
	hipComplex xu=out; 
	hipComplex yu=out;
	yu.r = z.i;
	int m,n;
	hipComplex ema=out;
	hipComplex ena=out;
	hipComplex den=out;
	for(m=-20;m<20;m++)
	{
		for(n=-20;n<20;n++)
		{
			if((m!=0)&&(n!=0))
			{
			if((gcd(m,n)==1))
			{
				ena.r = n;
				ema.r = m;
				den.r = norg(ema*z+ena);
				out = out + powc(yu,s)/powc(den,s/hav);
			}
			}
		}
	}
	return out;
}


__device__ hipComplex thu3(hipComplex z, hipComplex q)
{
    int v;
    hipComplex unity(1.0,0.0);
    hipComplex out(1.0,0.0);
    hipComplex tw(2.0,0.0);
    hipComplex qoo(1.0,0.0);
     for(v=0;v<10;v++)
    {
        qoo = qoo * q * q;
        out = out * (unity - qoo) * (unity + tw * qoo/q * asins(tw*z) + qoo*qoo/(q*q)); 
        
    }
    return out;
}

__device__ hipComplex trev(hipComplex lav, hipComplex mel, hipComplex rel)
{
hipComplex out(0.0,0.0);

hipComplex V(0.739085133215160641655312087674,0.0);
int v;
for(v=0;v<3;v++)
{
lav = lav - rel*(cosc(lav)-powc(V,rel))/cosc(lav);
out = out + mel*(cosc(lav)-powc(V,mel));
}

return out;
}

__device__ hipComplex polylog(hipComplex z, hipComplex s)
{
hipComplex out(0.0,0.0);
hipComplex oom(1.0,0.0);
hipComplex flag=oom;
int v;
for(v=0;v<30;v++)
{
    flag = flag + oom;
out = out + powc(z,flag)/powc(flag,s);
}

return out;
}

__device__
unsigned char clip(int n) { return n > 255 ? 255 : (n < 0 ? 0 : n); }

__global__
void distanceKernel(uchar4 *d_out, int w, int h, int2 pos) {
  const int c = blockIdx.x*blockDim.x + threadIdx.x;
  const int r= blockIdx.y*blockDim.y + threadIdx.y;
  const int i = c + r*w; // 1D indexing
  float pi = 3.1415926535898;
  hipComplex ip(pi,0.0);
    const float scale = 
    4.3;
    float fx = scale * (float)(DIM/2 - c)/(DIM/2);
    float fy = scale * (float)(DIM/2 - r)/(DIM/2);
    hipComplex effx(fx,0.0);
    hipComplex effy(fy,0.0);
    float LA = scale * (float)(DIM/2 - pos.x)/(DIM/2);
    float LB = scale * (float)(DIM/2 - pos.y)/(DIM/2);
    hipComplex mouse(LA,LB);
    hipComplex moux(LA,0.0);
    hipComplex mouy(0.0,LB);
    hipComplex q(fx,fy);

/*    hipComplex tik(sin(ticks/40.0f),0.0);*/
/*    hipComplex uon(cosf(-2*pi*ticks/16384.0),sinf(-2*pi*ticks/16384.0));
    hipComplex aon(cosf(2.6457513110645912*2*pi*ticks/1024),sinf(2.645751311064591*2*pi*ticks/1024));
    hipComplex eon(cosf(-2.6457513110645912*2*pi*ticks/1024.0),sinf(2.645751311064591*2*pi*ticks/1024.0));*/
        hipComplex fixon(.029348,.828934);
    hipComplex faxon(.029348,-.828934);
    hipComplex unity(1.0,0.0);
    hipComplex ai(0.0,1.0);
    
   hipComplex  aon = expc(ai*moux);
   hipComplex uon= expc(mouy);

    hipComplex flurn(0.0,0.0);
    hipComplex accume(1.0,0.0);
    hipComplex eccume(1.0,0.0);
    hipComplex rhun(1.02871376821872462237195122725097462534904479,0.0);
    hipComplex cue = q;
    hipComplex lam(0.73736887807831963, -0.67549029426152396);
    hipComplex due(3.0,0.0);
    hipComplex tir(2.0,0.0);
    hipComplex selga(3.5,0.0);


    hipComplex vro(-1.0,0.0);
    hipComplex tle(0.0,0.0);
    hipComplex sle(4.0,0.0);
    hipComplex cherra(0.62348980185873359, 0.7818314824680298);
    hipComplex lerra = cherra*cherra;
    hipComplex ferra = lerra * cherra;
    hipComplex terra = ferra * cherra;
    hipComplex zerra = terra * cherra;
    hipComplex nerra = zerra * cherra;
hipComplex vlarv(1/3.0,0.0);
    hipComplex sugna(0.70710678118654757, 0.70710678118654746);
    hipComplex regna(0.99966573338968745, 0.025853848581176047);
    hipComplex spa(sqrtf(2.0),0.0);
    hipComplex spb(sqrtf(3.0),0.0);
    hipComplex spc(sqrtf(4.0),0.0);
    hipComplex spd(sqrtf(5.0),0.0);
    hipComplex mrun(1/2.0,0.0);
hipComplex gloon (4.0,0.0);
    hipComplex plenod(-.01,0.0);
hipComplex nue = cue;
hipComplex bor(-10.0,0.0);
hipComplex nat(0.0,-10.0);
hipComplex rhus(1.0,0.0);
hipComplex D(0.739085133215160641655312087674,0.0);
hipComplex gren(2.0,0.0);
hipComplex next=flurn;
hipComplex current = cue;
hipComplex xnext = flurn;
hipComplex xcurrent = cue;
hipComplex rue=cue;
hipComplex tinny(.0001,0.0001);
hipComplex raga(0.5,27.0);
hipComplex ruga(0.5,0.0);
hipComplex senna(2.0,0.0);
hipComplex renna(3.0,0.0);
hipComplex finch(0.001,.001);
hipComplex smenn(0.5,sqrtf(3.0)/2.0);
hipComplex lmenn(0.96592582628906831,0.25881904510252074);
hipComplex vmenn(-0.5,-sqrtf(3.0)/2.0);
float ah, ex, feig;
feig = 	3.67;
ex = 2.10981;
float xa,xb,ya,yb,tta,ttb;
char va,vb,vc;
hipComplex seahorse(-0.75,0.123);

/*  if ((c >= w) || (r >= h)) return; // Check if within image bounds
  const int i = c + r*w; // 1D indexing
  const int dist = sqrtf((c - pos.x)*(c - pos.x) + 
                         (r - pos.y)*(r - pos.y));
  const unsigned char intensity = clip(255 - dist);*/
  
// theta function varying on constant
// cue =thess(cue,fixon*mouse);
int v=1;
int axa=-10;
int uu;


/*while((v<100)&&norg(cue)<2.0)
{
   cue = cue*(cue-mouy)*(cue-moux) -cue * q;
   v++;
}*/
   
// One way of describing this would be we want to perform Newton's method 
  //on the Mandelbrot set



/* preiterate */


//tex.stackexchange.com/questions/278843/making-a-phase-portrait-of-two-autonomous-system-of-differential-equations-with?fbclid=IwAR2Tz66CbUAq7LFVYck4uUGF5uQWnmzf5iZw3Bi8IOycvCC7czO6ZVgkz3s
// this is not terribly hard to do with cuda
   // what we need:
   // x' = x - y -> dx / dt = x - y
   // y' = 1 - x^2 -> dy / dt = 1-x^2
   // dy / dx = (dy / dt) / (dx/ dt)
   // so the trick is to convert dy/dx into a unit complex number to make this work, okay that's not that difficult

cue = mob(mouse,cue);
for(v=0;v<20;v++)
{
    cue = cue - mob(cue,mouse/mob(mouse,cue));
}









{

    double tha;
    tha = ((atan2(cue.i,cue.r) - pi)/(2.0*pi));

  
   d_out[i].x   = (unsigned char) (255.0*pow(sin(pi*tha),2));
    d_out[i].y = (unsigned char) (255.0*pow(sin(pi*tha+pi/3),2));
    d_out[i].z  = (unsigned char) (255.0*pow(sin(pi*tha+2*pi/3),2));
    d_out[i].w = 255;
 
   



    
}


}


 


void kernelLauncher(uchar4 *d_out, int w, int h, int2 pos) {
  const dim3 blockSize(TX, TY);
  const dim3 gridSize = dim3((w + TX - 1)/TX, (h + TY - 1)/TY);
  distanceKernel<<<gridSize, blockSize>>>(d_out, w, h, pos);
}

/*for(v=1;v<5;v++)
{
cue = cue - cue * (expc(unity-cue/moux)+expc(cue-unity/mouy))/((vlarv-unity/moux )*(expc(unity-cue/moux))-expc(cue-unity/mouy));
accume = accume + ((vlarv-unity/moux )*(expc(unity-cue/moux))-expc(cue-unity/mouy));
}

cue = accume;*/
/*cue = ramchi(moeb(unity,uon*fixon,q))*rampsi(moeb(unity,uon*fixon,q));
rhus = ramchi(uon/moeb(unity,uon*faxon,unity/q))*ramphi(uon/moeb(unity,uon*faxon,unity/q));
cue = rhus+cue;
cue = cosc(unity/(unity-uon*cue))*rampsi(moeb(unity,uon*fixon,q));*/


/*for(v=0;v<60;v++){
        cue = moeb(aon,fixon,cue) - aon/((expc(uon*cue-sins(cue))-cue)/((aon+cosc(cue)) * expc(uon*cue-sins(cue))-aon));
        accume = accume *(unity - (expc(aon*moeb(uon,faxon,cue))-sins(moeb(aon,fixon,cue))-cue));
    }
    cue = accume;*/

/*
One for
(x+d)/cos(d) -cos(x)/d
Tungilipa

D = cos(D)

cos(sqrt(x*D))/D -1 = 0.0




The other for
cos(x)-x
Eripgrunna
*/